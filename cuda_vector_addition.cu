#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c)
{
	for (int i=0; i<N; i++)
		c[i] = a[i] + b[i];
}

__global__ void device_add(int *a, int *b, int *c)
{
	// Thread + Block
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];

	// Thread only
	// c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

	// Block only
	//c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void gpu_alloc(void **devPtr, size_t size)
{
	hipError_t result = hipMalloc(devPtr, size);
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(result));
		exit(1);
	}
}

void gpu_free(void *devPtr)
{
	hipError_t result = hipFree(devPtr);
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(result));
	}
}

void cpu_to_gpu(void *devPtr, const void *hostPtr, size_t size)
{
	hipError_t result = hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		fprintf(stderr, "CPU -> GPU hipMemcpy failed: %s\n", hipGetErrorString(result));
	}
}

void gpu_to_cpu(void *hostPtr, const void *devPtr, size_t size)
{
	hipError_t result = hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		fprintf(stderr, "GPU -> CPU hipMemcpy failed: %s\n", hipGetErrorString(result));
	}
}

void fill(int *data)
{
	for (int i=0; i<N; i++)
		data[i] = i;
}

void print(int *a, int *b, int *c)
{
	for (int i=0; i<N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
}

int main()
{
	int *a, *b, *c;
	int *d_a = NULL, *d_b = NULL, *d_c = NULL;
	int allocation_size = N * sizeof(int);

	a = (int *)malloc(allocation_size);
	fill(a);

	b = (int *)malloc(allocation_size);
	fill(b);

	c = (int *)malloc(allocation_size);

	gpu_alloc((void **)&d_a, allocation_size);
	gpu_alloc((void **)&d_b, allocation_size);
	gpu_alloc((void **)&d_c, allocation_size);

	cpu_to_gpu(d_a, a, allocation_size);
	cpu_to_gpu(d_b, b, allocation_size);

	int threads_per_block = 128;
	int no_of_blocks = (N + threads_per_block - 1) / threads_per_block;

	device_add <<< no_of_blocks, threads_per_block >>> (d_a, d_b, d_c);

	gpu_to_cpu(c, d_c, allocation_size);

	print(a, b, c);

	free(a);
	free(b);
	free(c);

	gpu_free(d_a);
	gpu_free(d_b);
	gpu_free(d_c);

	return 0;
}

